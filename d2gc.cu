#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

__global__ colorWork(int* row_ptr, int* col_ind, int* neighbor_colors, int* color_array, int index, int i)
{
	int tid = 
	for(int j=row_ptr[i]; j < row_ptr[i+1]; j++){
		// neighborhood function to check neighbors of visited verte
		if(color_array[col_ind[j]] != -1){
			bool found = false;
			for(int c=0; c < index; c++){
				if(neighbor_colors[nov*tid+c] == color_array[col_ind[j]]){
					found = true;
					c = index;
				}
			}
			if(found == false){
				neighbor_colors[nov*tid+index] = color_array[col_ind[j]];
				index = index+1;
			}
			//printf("forbidden added first degree %d %d \n",col_ind[j],color_array[col_ind[j]]);
		}
		for(int k=row_ptr[col_ind[j]]; k < row_ptr[col_ind[j]+1];k++){
			if(color_array[col_ind[k]] != -1 && col_ind[k] != i){
				//colorı neighbor color içinde ara yoksa ekle
				bool alreadyIn = false;
				for(int c=0; c < index; c++){
					if(neighbor_colors[nov*tid+c] == color_array[col_ind[k]]){
						alreadyIn = true;
						c = index;
					}
				}
				if(alreadyIn == false){
					neighbor_colors[nov*tid+index] = color_array[col_ind[k]];
					index = index+1;
					//printf("forbidden added second degree %d %d \n",col_ind[k],color_array[col_ind[k]]);
				}
			}
		}
	}
}

void callD2GC(int* row_ptr, int* col_ind, int nov)
{
	float totaltime;
  hipEvent_t startEvent, endEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&endEvent);
  // 
	int *color_array, *d_color_array, d_neighbor_colors;//array to keep colors of the vertices the color numbers start from 1
 	__shared__ int *d_neighbor_colors;
  hipMalloc( (void**)&color_array, nov*sizeof(int));
  hipMalloc( (void**)&neighbor_colors, nov*sizeof(int));
  hipMalloc( (void**)&d_color_array, nov*sizeof(int));
  hipMalloc( (void**)&d_neighbor_colors, nov*sizeof(int));
  hipMemset( color_array, -1, nov*sizeof(int));
//	neighbor_colors =(int*)malloc(thread_num*nov*sizeof(int));
	hipMemcpy(d_color_array, color_array, nov*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbor_colors, neighbor_colors, nov*sizeof(int), hipMemcpyHostToDevice);

	int maxColor;//to print out color number
	typedef int bool;//to boolean check
	enum {false, true};
	printf("Nov is %d\n", nov);
	printf("Started...");
	
	bool isFinished = false;//to check all the vertices colored without conflict and coloring finished	
	hipEventRecord(startEvent, 0);
	while(!isFinished){		
		printf("Turn ");
		isFinished = true;//if it is not finished it is changed in conflict check part
		//COLORWORKQUEUE			
		#pragma omp parallel for num_threads(thread_num) shared(color_array)
		for(int i=0; i < nov; i++){//in parallel visit all the vertices in graph
			int tid = omp_get_thread_num();
			if(color_array[i] == -1){//checks if vertex is colored in previous turns
				int index = 0;//keeps number of colored neighbors
				//forbidden color bulma
				colorWork<<1,1>>(row_ptr, col_ind, );
				
				/*printf("vertex is %d\nForbidden of %d\n",i,i);
				for(int k=0;k<index;k++){
					printf("%d ",neighbor_colors[nov*tid+k]);
				}
				printf("\n");*/
				int col = 0;
				bool sameWithNbor = true;
				while(sameWithNbor){
					sameWithNbor = false;
					for(int k=0; k < index; k++){
						if(col == neighbor_colors[nov*tid+k]){
							col = col+1;
							sameWithNbor = true;
						}
					}
				}
				color_array[i] = col;
			}
		}
		// REMOVECONFLICTS
		//TODO: Check d2 vertices
		
		#pragma omp parallel for num_threads(thread_num) shared(isFinished, color_array)
		for(int i=0; i < nov; i++){
			for(int j=row_ptr[i]; j < row_ptr[i+1]; j++){
				if(color_array[col_ind[j]] == color_array[i] && i > col_ind[j]){//if neighbor and vertex have same color and index of vertex is greater than neighbor
					color_array[i] = -1;
					j = row_ptr[i+1];
					isFinished = false;
				}
				if(isFinished == true){
					for(int k= row_ptr[col_ind[j]]; k < row_ptr[col_ind[j]+1]; k++){
						if(color_array[col_ind[k]] == color_array[i] && i > col_ind[k]){
							color_array[i] = -1;
							k = row_ptr[col_ind[j]+1];
							j = row_ptr[i+1];
							isFinished = false;
						}
					}
				}
			}
		}/*
		printf("Colors are:\n");
		for(int i=0; i<nov;i++){
	    printf("%d  ",color_array[i]);			
		}*/
		//printf("\n");
	}
	
	printf("\n");
	hipEventRecord(endEvent, 0);
  hipEventSynchronize(endEvent);
	hipEventElapsedTime(&totaltime, startEvent, endEvent);
  printf("Execution time is %f secs.\n", totaltime/1000);

	maxColor = color_array[0];
	//printf("%d  ",color_array[0]);		
	for(int i=1; i<nov;i++){
	    //printf("%d  ",color_array[i]);
		if(maxColor < color_array[i]){
			maxColor = color_array[i];
		}
	}
	printf("\nNumber of colors is %d\n", maxColor+1);

	char result_name[1024];
	strcpy(result_name,"resultOf-");
	char mtx_name[255];
	int index = strstr(fname,".")- fname;
	strncpy(mtx_name, fname, index);
	mtx_name[index] = '\0';
	sprintf(result_name,"%s%s%s",result_name, mtx_name, ".txt");

	FILE *f = fopen(result_name, "w");
	if(f == NULL){
		printf("Cannot open result_file to write\n");
		exit(1);
	}
	fprintf(f,"%d", maxColor+1);
	fprintf(f,"\n");
	for(int i = 0; i<nov;i++){
		fprintf(f,"%d ",color_array[i]);
	}
	fclose(f);

	hipFree(color_array);
	hipFree(neighbor_colors);
}
