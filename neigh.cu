
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <limits.h>

__global__ void operate(int *test, int *train, double *dist, int tr_num, int index, int dimen){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

 
  //printf("%d  %d\n", tid, tr_num);
  if(tid < tr_num)
  {
  	double sum = 0.0;
/* 	
    int loc = tid*dimen;
  	sum =  (train[loc+0] - test[0+index])  *  (train[loc+0] - test[0+index]) +
  				 (train[loc+1] - test[1+index])  *  (train[loc+1] - test[1+index]) +
  				 (train[loc+2] - test[2+index])  *  (train[loc+2] - test[2+index]) +
  				 (train[loc+3] - test[3+index])  *  (train[loc+3] - test[3+index]) +
  				 (train[loc+4] - test[4+index])  *  (train[loc+4] - test[4+index]) +
  				 (train[loc+5] - test[5+index])  *  (train[loc+5] - test[5+index]) +
  				 (train[loc+6] - test[6+index])  *  (train[loc+6] - test[6+index]) +
  				 (train[loc+7] - test[7+index])  *  (train[loc+7] - test[7+index]) +
  				 (train[loc+8] - test[8+index])  *  (train[loc+8] - test[8+index]) +
  				 (train[loc+9] - test[9+index])  *  (train[loc+9] - test[9+index]) +
  				(train[loc+10] - test[10+index]) * (train[loc+10] - test[10+index]) +
  				(train[loc+11] - test[11+index]) * (train[loc+11] - test[11+index]) +
  				(train[loc+12] - test[12+index]) * (train[loc+12] - test[12+index]) +
  				(train[loc+13] - test[13+index]) * (train[loc+13] - test[13+index]) + 
  				(train[loc+14] - test[14+index]) * (train[loc+14] - test[14+index]) +
  				(train[loc+15] - test[15+index]) * (train[loc+15] - test[15+index]);
  	
  	*/
  	for(int i = 0; i < dimen; i++){
  		sum = sum + (train[tid*dimen+i] - test[i+index]) * (train[tid*dimen+i] - test[i+index]); 
  	}
  	
  	dist[tid] = sum;
    //printf("%d : %lf\n", tid,sum);
  }
}

__global__ void write(int *bla, int size)
{
 for (int i = 0; i < size; ++i)
  {
    if(i != 0 && i%16 == 0) printf("\nline %d \n", i/16);
    printf("%d ", bla[i]);
  }
}

void call(int *test, int *train, double *dist, int *d_test, int *d_train, double *d_dist, int ts_num, int tr_num, int dimen)
{
  int ts_size = ts_num*dimen;
  int tr_size = tr_num*dimen;
  
//  printf("%d\n", ts_size);
//  printf("%d\n", tr_size);
 

  hipMalloc( (void**)&d_test, ts_size*sizeof(int));
  hipMalloc( (void**)&d_train, tr_size*sizeof(int));
  hipMalloc( (void**)&d_dist, tr_num*sizeof(double));

  hipMemcpy(d_test, test, ts_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_train, train, tr_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_dist, dist, tr_num*sizeof(double), hipMemcpyHostToDevice);
  //write<<<1,1>>>(d_test, ts_size);
  //write<<<1,1>>>(d_train, tr_size);

  FILE *f = fopen("out.txt", "w");
	if (f == NULL)
	{
    printf("Error opening file!\n");
    exit(1);
	}
  const int blockSize = 20;
  const int bla = 1024;

  float totaltime;
  hipEvent_t startEvent, endEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&endEvent);
  hipEventRecord(startEvent, 0);

  for(int i = 0; i < ts_size; i+=dimen)
  {
    
    operate<<<blockSize, bla>>>(d_test, d_train, d_dist, tr_num, i, dimen);
    hipMemcpy(dist, d_dist, tr_num*sizeof(double), hipMemcpyDeviceToHost);

    double min_dist = 100000000.0;
    int which = -1;
    for(int j = 0; j<tr_num; j++)
    {
      if(min_dist>dist[j]){
        which = j;
        min_dist = dist[j];
      }
    }
    //printf("test: %d,\ttrain: %d,\tdistance: %lf\n", i/16, which, sqrt(min_dist));
    fprintf(f, "%d\n", which);
  }	
  hipEventRecord(endEvent, 0);
  hipEventSynchronize(endEvent);
	hipEventElapsedTime(&totaltime, startEvent, endEvent);
	fclose(f);
  printf("Execution time is %f secs.\n", totaltime/1000);

  hipFree(d_test);
  hipFree(d_train);
  hipFree(d_dist);

}